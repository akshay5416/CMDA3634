#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


   
__device__ unsigned int modProd(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int modExponential(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modProd(aExpb, z, p);
    z = modProd(z, z, p);
    b /= 2;
  }
  return aExpb;
}

// find the secret key
__global__ void findSecretKey(unsigned int g, unsigned int p, unsigned int h, unsigned int *d_a){
  int thread = threadIdx.x;
  int block  = blockIdx.x;
  int blockSize = blockDim.x;

  //unique global thread ID
  int id = thread + block*blockSize;

 if (id < p-1) {  
   // for (unsigned int i=0;i<p-1;i++) {
      if (modExponential(g, id,p)==h) {
       //iintf("Secret key found! x = %u \n", i+1);
        *d_a=id;  
      }
    }
   // double endTime = clock();

   // double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
   // double work = (double) p;
   // double throughput = work/totalTime;

   // printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }



int main (int argc, char **argv) {


  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

//declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
    FILE* file;
    file = fopen("public_key.txt", "r");
    fscanf(file,"%u\n", &n);
    fscanf(file,"%u\n", &p);
    fscanf(file,"%u\n", &g);
    fscanf(file,"%u\n", &h);
    fclose(file);

    FILE *file2;
    file2 = fopen("messages.txt", "r");
    unsigned int count;
    fscanf(file2,"%u\n", &count);
    Nints = count;
    unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
    unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));
    for(int i = 0; i<count; i++){
        fscanf(file2, "%u %u\n", &Zmessage[i], &a[i]);
    }
    fclose(file2);

/* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
    // find the secret key 
    int Nthreads = 32;
    dim3 B(32,1,1);
    //grid dimensions
    dim3 G((p+32-1)/32,1,1);

    unsigned int *d_a; 
    unsigned int *h_a; 

    hipMalloc(&d_a, Nthreads*sizeof(unsigned int));
    h_a = (unsigned int *) malloc(sizeof(unsigned int));

    double deviceStart = clock();
    findSecretKey <<<G,B >>>(g,p, h,d_a);
    hipDeviceSynchronize();

    double deviceEnd = clock();
    double deviceTime = (deviceEnd-deviceStart)/(double) CLOCKS_PER_SEC;
    
    hipMemcpy(h_a, d_a, sizeof(unsigned int),hipMemcpyDeviceToHost);

    printf("The secret key is %u \n", *h_a);


    /* Q3 After finding the secret key, decrypt the message */
  int bufferSize = 1024;
  print("This is error 1 \n");
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  print("This is error 2 \n");
  unsigned int charsPerInt = (n-1)/8 ;
  print("This is error3 \n");
  unsigned int Nchars = mystrlen(message);
  print("This is error 4 \n");
  Nints = mystrlen(message)/charsPerInt;
  print("This is error 5 \n");
  ElGamalDecrypt(Zmessage,a,Nints,p,x);

  convertZToString(Zmessage, Nints, message, Nchars);
  printf("Decrypted Message = %s\n", message);

  hipFree(d_a);
  free(h_a);

  return 0;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"


   
__device__ unsigned int modProd(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int modExponential(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modprod(aExpb, z, p);
    z = modprod(z, z, p);
    b /= 2;
  }
  return aExpb;
}

// find the secret key
__global__ void int findSecretKey(unsigned int g, unsigned int p, unsigned int h, unsigned int *d_a){
  int thread = threadIdx.x;
  int block  = blockIdx.x;
  int blockSize = blockDim.x;
  int gridSize  = gridDim.x;

  //unique global thread ID
  int id = thread + block*blockSize;

 if (id < p-1) {  
   // for (unsigned int i=0;i<p-1;i++) {
      if (modExponential(g, id,p)==h) {
       //iintf("Secret key found! x = %u \n", i+1);
        d_a=id+1;
      }
    }
   // double endTime = clock();

   // double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
   // double work = (double) p;
   // double throughput = work/totalTime;

   // printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }



int main (int argc, char **argv) {


  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */
    //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */
      int bufferSize = 1024;
      unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
      unsigned int charsPerInt = (n-1)/8 ;
      unsigned int Nchars = strlen(message);
      Nints = strlen(message)/charsPerInt;


    FILE* file;
    file = fopen("public_key.txt", "r");
    fscanf(file,"%u\n", &n);
    fscanf(file,"%u\n", &p);
    fscanf(file,"%u\n", &g);
    fscanf(file,"%u\n", &h);
    fclose(file);

     FILE *file2;
    file2 = fopen("messages.txt", "r");
    unsigned int count;
    fscanf(file2,"%u\n", &count);
    unsigned int *Zmessage = (unsigned int *) malloc(Nints*sizeof(unsigned int));
    unsigned int *a = (unsigned int *) malloc(Nints*sizeof(unsigned int));
    for(int i = 0; i<count; i++){
        fscanf(file2, "%u %u\n", &Zmessage[i], &a[i]);
    }
    fclose(file2);
  // find the secret key
 // if (x==0 || modExp(g,x,p)!=h) {
   // printf("Finding the secret key...\n");
   // double startTime = clock();
   // for (unsigned int i=0;i<p-1;i++) {
     // if (modExp(g,i+1,p)==h) {
       // printf("Secret key found! x = %u \n", i+1);
       // x=i+1;
      }
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  }
    
    /* Q3 After finding the secret key, decrypt the message */
  int bufferSize = 1024;
  unsigned char *message = (unsigned char *) malloc(bufferSize*sizeof(unsigned char));
  unsigned int charsPerInt = (n-1)/8 ;
  unsigned int Nchars = strlen(message);
  Nints = strlen(message)/charsPerInt;
  ElGamalDecrypt(Zmessage,a,Nints,p,x);

  convertZToString(Zmessage, Nints, message, Nchars);
  printf("Decrypted Message = %s\n", message);

/* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
  int Nthreads = 32; 
  dim3 B(32, 1, 1);
  dim3 G((p-1+32)/32,1,1);

  double deviceStart = clock();

  float *h_a = (float*) malloc(sizeof(float));
  float *d_a; 
  hipMalloc(&d_a, Nthreads*sizeof(float));

  hipMemcpy(d_a,h_a,Nthreads*sizeof(float),hipMemcpyHostToDevice);  
  
  findSecretKey<<< G,B >>> (g, p, h,*d_a);
  hipDeviceSynchronize();

  double deviceEnd = clock();
  double deviceTime = (deviceEnd-deviceStart)/(double) CLOCKS_PER_SEC;
  
   hipMemcpy(h_a,d_a, Nthreads*sizeof(float), hipMemcpyDeviceToHost);

  printf("The secret key is %f\n ", h_a);
 // printf("The device took %f seconds to add a and b \n", deviceTime); 
 // printf("The effective bandwith of the device was % GB/s\n", totalMem/(1E9*deviceTime));
  
  hipFree(d_a);
  free(h_a);
  return 0;
}
